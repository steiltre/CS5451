#include "hip/hip_runtime.h"

#define THREADSPERBLOCK 512
#define WARPLENGTH 32

#include <stdio.h>
#include <math.h>

extern "C"
{
#include "image.h"
#include "stencil.h"
}

__device__ __constant__ float d_stencil[9];

__global__ void cuda_apply_stencil(
        float * input,
        float * output,
        int width,
        int height)
{
    extern __shared__ float tile[];  /* Tile needed by block */

    float temp;
    temp = 0;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    /* Fetch tile values from global memory and place in shared memory (need extra column and row around boundary for applying stencil) */
    if (row > 0 && col > 0 && row < height+1 && col < width+1) {
        tile[ threadIdx.y * (blockDim.x+2) + threadIdx.x ] = input[ (row-1)*width + (col-1) ];
    }

    if (threadIdx.x < 2 && row > 0 && row < height+1 && col + blockDim.x < width) {
        tile[ threadIdx.y * (blockDim.x+2) + threadIdx.x + blockDim.x ] = input[ (row-1)*width + col + blockDim.x - 1 ];
    }

    if (threadIdx.y < 2 && col > 0 && col < width+1 && row + blockDim.y < height) {
        tile[ (threadIdx.y + blockDim.y) * (blockDim.x+2) + threadIdx.x ] = input[ (row + blockDim.y - 1)*width + (col-1) ];
    }

    if (threadIdx.x < 2 && threadIdx.y < 2 && col + blockDim.x < width && row + blockDim.y < height) {
        tile[ (threadIdx.y + blockDim.y) * (blockDim.x+2) + threadIdx.x + blockDim.x ] = input[ (row + blockDim.y - 1)*width + col + blockDim.x - 1 ];
    }

    __syncthreads();

    if (row >= 0 && col >= 0 && row < height && col < width) {
        if (row == 0 || col == 0 || row == height-1 || col == width-1) {
            temp = tile[ (threadIdx.y+1) * (blockDim.x+2) + threadIdx.x+1 ];
            //temp = input[ row * width + col ];
        }
        else {
            for (int i=0; i<3; i++) {
                for (int j=0; j<3; j++) {
                    temp += d_stencil[i*3 + j] * tile[ (threadIdx.y+i)*(blockDim.x+2) + threadIdx.x+j ];
                }
            }
        }

        output[ row*width + col ] = temp;
    }

}

image_t * stencil_cuda(
    image_t const * const input,
    float stencil[3][3],
    int const num_times)
{
  image_t * output = image_alloc(input->width, input->height);

  int arr_size = input->width*input->height;

  float *d_redout, *d_greenout, *d_blueout, *d_redin, *d_greenin, *d_bluein;

  hipMalloc( (void**) &d_redout, 3*arr_size * sizeof(*d_redout) );
  //hipMalloc( (void**) &d_greenout, arr_size );
  //hipMalloc( (void**) &d_blueout, arr_size );
  hipMalloc( (void**) &d_redin, 3*arr_size  * sizeof(*d_redin) );
  //hipMalloc( (void**) &d_greenin, arr_size );
  //hipMalloc( (void**) &d_bluein, arr_size );

  d_greenout = d_redout + arr_size;
  d_blueout = d_redout + 2*arr_size;
  d_greenin = d_redin + arr_size;
  d_bluein = d_redin + 2*arr_size;

  hipMemcpyToSymbol(HIP_SYMBOL(d_stencil), stencil, 9*sizeof(float));

  hipMemcpy(d_redin, input->red, 3*arr_size * sizeof(*d_redin) , hipMemcpyHostToDevice);
  //hipMemcpy(d_greenin, input->green, arr_size, hipMemcpyHostToDevice);
  //hipMemcpy(d_bluein, input->blue, arr_size, hipMemcpyHostToDevice);

  /* Determine dimensions of blocks and grid */
  /* Want blocks that are roughly square but also have length that is a multiple of WARPLENGTH for coalescing */
  int block_len =  ( (int) ceil( sqrt( float(THREADSPERBLOCK) ) / WARPLENGTH ) ) * WARPLENGTH;
  dim3 block( block_len, THREADSPERBLOCK / block_len );
  dim3 grid( ceil( input->width/float(block.x) ), ceil( input->height/float(block.y) ) );

  for (int i=0; i < num_times; ++i) {
      /* Apply stencil to each channel separately. */
      cuda_apply_stencil<<<grid, block, (block.x+2)*(block.y+2)*sizeof(float)>>>(d_redin, d_redout, input->width, input->height);
      cuda_apply_stencil<<<grid, block, (block.x+2)*(block.y+2)*sizeof(float)>>>(d_greenin, d_greenout, input->width, input->height);
      cuda_apply_stencil<<<grid, block, (block.x+2)*(block.y+2)*sizeof(float)>>>(d_bluein, d_blueout, input->width, input->height);
  }

  hipMemcpy(output->red, d_redout, 3 * arr_size * sizeof(*d_redout), hipMemcpyDeviceToHost);
  //hipMemcpy(output->green, d_greenout, arr_size, hipMemcpyDeviceToHost);
  //hipMemcpy(output->blue, d_blueout, arr_size, hipMemcpyDeviceToHost);

  hipFree(d_redout);
  //hipFree(d_greenout);
  //hipFree(d_blueout);
  hipFree(d_redin);
  //hipFree(d_greenin);
  //hipFree(d_bluein);

  return output;
}


